//
//

#include <benchmark/benchmark.h>
#include <thrust/device_vector.h>

#include "bench_config.cuh"
#include "small_batched_gemm_pipeline.cuh"


inline constexpr auto rm_kernel = small_batched_cooperative_pipelined_gemm<float, 6, 2, 256, Eigen::RowMajor>;
inline constexpr auto cm_kernel = small_batched_cooperative_pipelined_gemm<float, 6, 2, 256, Eigen::ColMajor>;

static void bench_small_batched_cooperative_gemm_pipelined_6x6_rm(benchmark::State &state) {
    constexpr int dim = 6;
    constexpr int size = dim*dim;
    auto n_matrices = static_cast<int>(state.range(0));

    thrust::device_vector<float> a(size * n_matrices, 1.0f);
    thrust::device_vector<float> b(size * n_matrices, 1.0f);
    thrust::device_vector<float> c(size * n_matrices, 0.0f);

    for (auto _: state) {

        const auto threads = 256;
        // TODO: This is over-estimating the number of blocks that are actually needed.
        const auto blocks = (n_matrices + threads - 1) / threads;
        float alpha = 1.0;
        float beta = 0.0;

        rm_kernel<<<blocks, threads>>>(
            raw_pointer_cast(a.data()),
            raw_pointer_cast(b.data()),
            raw_pointer_cast(c.data()),
            alpha,
            beta,
            n_matrices);

        hipDeviceSynchronize();
    }
}

BENCHMARK(bench_small_batched_cooperative_gemm_pipelined_6x6_rm)->Arg(kNumMatrices);