#include <benchmark/benchmark.h>

#include "recursive_matrix.cuh"


int main(int argc, char **argv) {
    benchmark::MaybeReenterWithoutASLR(argc, argv);
    char arg0_default[] = "benchmark";
    char *args_default = reinterpret_cast<char *>(arg0_default);
    if (!argv) {
        argc = 1;
        argv = &args_default;
    }
    ::benchmark::Initialize(&argc, argv);
    if (::benchmark::ReportUnrecognizedArguments(argc, argv)) return 1;
    ::benchmark::RunSpecifiedBenchmarks();
    ::benchmark::Shutdown();

    hipDeviceReset();
    return 0;
}
