//
//

#include <benchmark/benchmark.h>
#include <thrust/device_vector.h>

#include "bench_config.cuh"
#include "tiny_batched_gemm_cls.cuh"



inline constexpr auto tiny_batched_gemm_3x3_rm_cls = tiny_batched_gemm_cls<float, 3, 256, Eigen::RowMajor>;

static void bench_tiny_batched_gemm_3x3_rm_cls(benchmark::State &state) {
    constexpr int dim = 3;
    constexpr int size = dim * dim;
    const auto n_matrices = static_cast<int>(state.range(0));


    thrust::device_vector<float> a(size * n_matrices);
    thrust::device_vector<float> b(size * n_matrices);
    thrust::device_vector<float> c(size * n_matrices);

    for (auto _: state) {
        const auto threads = 256;
        const auto blocks = (n_matrices + threads - 1) / threads;

        float alpha = 1.0;
        float beta = 0.0;


        tiny_batched_gemm_3x3_rm_cls<<<blocks, threads>>>(
            raw_pointer_cast(a.data()),
            raw_pointer_cast(b.data()),
            raw_pointer_cast(c.data()),
            alpha,
            beta,
            n_matrices);


        hipDeviceSynchronize();
    }
}

BENCHMARK(bench_tiny_batched_gemm_3x3_rm_cls)->Arg(kNumMatrices);
